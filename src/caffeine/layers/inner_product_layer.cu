#include "hip/hip_runtime.h"
#include <mkl.h>
#include <hipblas.h>

#include "caffeine/blob.hpp"
#include "caffeine/common.hpp"
#include "caffeine/filler.hpp"
#include "caffeine/layer.hpp"
#include "caffeine/vision_layers.hpp"

namespace caffeine{

    template <typename Dtype>
    void InnerProductLayer<Dtype>::SetUp(const vector<Blob<Dtype>* >& bottom, vector<Blob<Dtype>* >* top){
        CHECK_EQ(bottom.size(), 1) << "IP Layer takes a single blob as input.";
        CHECK_EQ(top->size(), 1) << "IP Layer takes a single blob as output. ";
        const int num_output = this->layer_param_.num_output();
        const bool gemm_last_dim = this->layer_param_.gemm_last_dim();
        biasterm_ = this->layer_param_.bias_term();
        // Figure out the dimemsions
        if(gemm_last_dim){
            M_ = bottom[0]->count() / bottom[0]->channels();
            K_ = bottom[0]->channels();
            N_ = num_output;
            (*top)[0]->Reshape(bottom[0]->num(), bottom[0]->height(),
                               bottom[0]->width(), num_output);
        } else {
            M_ = bottom[0]->num();
            K_ = bottom[0]->count() / bottom[0]->num();
            N_ = num_output;
            (*top)[0]->Reshape(bottom[0]->num(), 1, 1, num_output);
        }

        if (biasterm_) {
            this->blobs_.resize(2);
        } else {
            this->blobs_.resize(1);
        }
        // Intialize the weight
        this->blobs_[0].Reshape(1, 1, K_, N_);
        // fill the weights
        shared_ptr<Filler<Dtype> > weight_filler(
                GetFiller<Dtype>(this->layer_param_.weight_filler()));
        weight_filler->Fill(&(this->blobs_[0]));

        // If necessary, intiialize and fill the bias term
        if (biasterm_) {
            this->blobs_[1].Reshape(1, 1, 1, N_);
            shared_ptr<Filler<Dtype> > bias_filler(
                    GetFiller<Dtype>(this->layer_param_.bias_filler()));
            bias_filler->Fill(&(this->blobs_[1]));
            bias_multiplier_.reset(new SyncedMemory(M_ * sizeof(Dtype)));
            Dtype* bias_multiplier_data = (Dtype*)bias_multiplier_->mutable_cpu_data();
            for (int i = 0; i < M_; ++i) {
                bias_multiplier_data[i] = 1.;
            }
        }
    } // setup fun

    template <typename Dtype>
    void InnerProductLayer<Dtype>::Forward_cpu(const vector<Blob<Dtype>*>& bottom,
                                               vector<Blob<Dtype>*>* top) {
        const Dtype* bottom_data = bottom[0]->cpu_data();
        Dtype* top_data = (*top)[0]->mutable_cpu_data();
        const Dtype* weight = this->blobs_[0].cpu_data();
        const Dtype* bias = NULL;
        if (biasterm_) {
            bias = this->blobs_[1].cpu_data();
        }
        switch(sizeof(Dtype)) {
            case sizeof(float):
                // matrix multiply
                cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, M_, N_, K_,
                            1., (const float*)bottom_data, K_, (const float*)weight, N_, 0.,
                            (float*)top_data, N_);
                if (bias) {
                    // add bias
                    cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, M_, N_, 1,
                                1., (const float*)bias_multiplier_->cpu_data(), 1,
                                (const float*)bias, N_, 1., (float*)top_data, N_);
                }
                break;
            case sizeof(double):
                // matrix multiply
                cblas_dgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, M_, N_, K_,
                            1., (const double*)bottom_data, K_, (const double*)weight, N_, 0.,
                            (double*)top_data, N_);
                if (bias) {
                    // add bias
                    cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, M_, N_, 1,
                                1., (const float*)bias_multiplier_->cpu_data(), 1,
                                (const float*)bias, N_, 1., (float*)top_data, N_);
                }
                break;
            default:
                CHECK(false) << "Unknown data type.";
        }
    } // Forward_cpu fun

    template <typename Dtype>
    Dtype InnerProductLayer<Dtype>::Backward_cpu(const vector<Blob<Dtype>* >& top,
                                                 const bool propagate_down,
                                                 vector<Blob<Dtype>*>* bottom) {
        // TODO: gradient w.r.t the params
        if (propagate_down) {
            // TODO: gradient w.r.t. the bottom
        }
        return Dtype(0);
    }// Baceward_cpu fun

    template <typename Dtype>
    __global__ void BroadcastRow(const int total, const int vec_len,
                                  const Dtype* in_vec, Dtype* out_matrix) {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        if (index < total) {
            int v_index = index % vec_len;
            out_matrix[index] = in_vec[v_index];
        }
    }

    template <typename Dtype>
    void InnerProductLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                               vector<Blob<Dtype>*>* top) {
        const Dtype* bottom_data = bottom[0]->gpu_data();
        Dtype* top_data = (*top)[0]->mutable_gpu_data();
        const Dtype* weight = this->blobs_[0].gpu_data();
        const Dtype* bias = NULL;
        Dtype alpha = 1., beta = 0.;
        if (biasterm_) {
            bias = this->blobs_[1].gpu_data();
            beta = 1.;
            const int count = (*top)[0]->count();
            // we pre-copy the bias to the results, and then call gemm.
            BroadcastRow<<<CAFFEINE_GET_BLOCKS(count), CAFFEINE_CUDA_NUM_THREADS>>>(
                    count, N_, bias, top_data);
        }
        switch(sizeof(Dtype)) {
            case sizeof(float):
                // matrix multiply: since cublas uses Fortran major, we actually do
                // C' = B' A'
                CUBLAS_CHECK(hipblasSgemm(Caffeine::cublas_handle(), HIPBLAS_OP_N,
                                         HIPBLAS_OP_N, N_, M_, K_, (float*)&alpha, (const float*)weight, N_,
                                         (const float*)bottom_data, K_, (float*)&beta, (float*)top_data, N_));
                break;
            case sizeof(double):
                // matrix multiply
                CUBLAS_CHECK(hipblasDgemm(Caffeine::cublas_handle(), HIPBLAS_OP_N,
                                         HIPBLAS_OP_N, N_, M_, K_, (double*)&alpha, (const double*)weight, N_,
                                         (const double*)bottom_data, K_, (double*)&beta, (double*)top_data, N_));
                break;
            default:
                CHECK(false) << "Unknown data type.";
        }
    }

    template <typename Dtype>
    Dtype InnerProductLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                                 const bool propagate_down,
                                                 vector<Blob<Dtype>*>* bottom) {
        CHECK(false);
        return Dtype(0.);
    }

    INSTANTIATE_CLASS(InnerProductLayer);


} // namespace caffeine