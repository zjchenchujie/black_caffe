#include "hip/hip_runtime.h"
//
// Created by chujie on 1/14/19.
//
#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffeine/common.hpp"
#include "caffeine/util/im2col.hpp"

namespace caffeine {

    template <typename Dtype>
    __global__ void im2col_gpu_kernel(const int n, const Dtype* data_im,
                                      const int height, const int width, const int ksize,
                                      const int stride, const int height_col, const int width_col, Dtype* data_col) {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        if (index < n) {
            int w_out = index % width_col;
            index /= width_col;
            int h_out = index % height_col;
            int channel_in = index / height_col;
            int channel_out = channel_in * ksize * ksize;
            int h_in = h_out * stride;
            int w_in = w_out * stride;
            data_col += (channel_out * height_col + h_out) * width_col + w_out;
            data_im += (channel_in * height + h_in) * width + w_in;
            for (int i = 0; i < ksize; ++i) {
                for (int j = 0; j < ksize; ++j) {
                    *data_col = data_im[i * width + j];
                    data_col += height_col * width_col;
                }
            }
        }
    }

    template <typename Dtype>
    void im2col_gpu(const Dtype* data_im, const int channels,
                    const int height, const int width, const int ksize, const int stride,
                    Dtype* data_col) {
        // We are going to launch channels * height_col * width_col kernels, each
        // kernel responsible for copying a single-channel grid.
        int height_col = (height - ksize) / stride + 1;
        int width_col = (width - ksize) / stride + 1;
        int num_kernels = channels * height_col * width_col;
        im2col_gpu_kernel<<<CAFFEINE_GET_BLOCKS(num_kernels), CAFFEINE_CUDA_NUM_THREADS>>>(
                num_kernels, data_im, height, width, ksize, stride, height_col, width_col,
                        data_col);
        CUDA_POST_KERNEL_CHECK;
    }

// Explicit instantiation
    template void im2col_gpu<float>(const float* data_im, const int channels,
                                    const int height, const int width, const int ksize, const int stride,
                                    float* data_col);
    template void im2col_gpu<double>(const double* data_im, const int channels,
                                     const int height, const int width, const int ksize, const int stride,
                                     double* data_col);

/*
template <typename Dtype>
void col2im_gpu(const Dtype* data_col, const int channels,
    const int height, const int width, const int ksize, const int stride,
    Dtype* data_im) {
  memset(data_im, 0, sizeof(Dtype) * height * width * channels);
  int height_col = (height - ksize) / stride + 1;
  int width_col = (width - ksize) / stride + 1;
  int channels_col = channels * ksize * ksize;
  for (int c = 0; c < channels_col; ++c) {
    int w_offset = c % ksize;
    int h_offset = (c / ksize) % ksize;
    int c_im = c / ksize / ksize;
    for (int h = 0; h < height_col; ++h) {
      for (int w = 0; w < width_col; ++w) {
        data_im[(c_im * height + h * stride + h_offset) * width + w * stride
            + w_offset] += data_col[(c * height_col + h) * width_col + w];
      }
    }
  }
}

// Explicit instantiation
template void col2im_gpu<float>(const float* data_col, const int channels,
    const int height, const int width, const int psize, const int stride,
    float* data_im);
template void col2im_gpu<double>(const double* data_col, const int channels,
    const int height, const int width, const int psize, const int stride,
    double* data_im);
*/
}  // namespace caffeine

